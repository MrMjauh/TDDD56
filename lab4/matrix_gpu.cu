// Matrix addition, CPU version
// gcc matrix_cpu.c -o matrix_cpu -std=c99

/*
answers

N = 2**6 = 64 is the turning point in gpu v cpu
block size 16*16 gave the best performance for us

coalescing
on N=2**10
Time 0.4686
Time 0.1143
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void add_matrix(float *a, float *b, float *c, int N)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int index = x + y*N;
	if (index < N*N) // allow allocating more threads than elements
		c[index] = a[index] + b[index];
}


void add_matrix_cpu(float *a, float *b, float *c, int N)
{
	for (int y=0; y < N; y++)
		for (int x=0; y < N; x++){
			int index = x + y*N;
			c[index] = a[index] + b[index];
		}
}

int main()
{
	const int N = 1<<10;
	const int blockSize = 16;
	const int size = N*N*sizeof(float);
	float t;

	float *a, *ad;
	float *b, *bd;
	float *c, *cd;

	a = new float[N*N];
	b = new float[N*N];
	c = new float[N*N];
	hipMalloc( (void**)&ad, size );
	hipMalloc( (void**)&bd, size );
	hipMalloc( (void**)&cd, size );

	hipEvent_t event0, event1;
	hipEventCreate(&event0);
	hipEventCreate(&event1);

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}

	hipMemcpy( ad, a, size, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, size, hipMemcpyHostToDevice );

	dim3 dimBlock( blockSize, blockSize );
	dim3 dimGrid( N/blockSize, N/blockSize );

	hipEventRecord(event0, 0);
	add_matrix<<< dimGrid, dimBlock >>>(ad, bd, cd, N);
	hipEventRecord(event1, 0);

	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost );

	hipEventSynchronize(event0);
	hipEventSynchronize(event1);
	hipEventElapsedTime(&t, event0, event1);

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			if (i==N-1 && N-32 < j)printf("%0.2f ", c[i+j*N]);
		}
		//printf("\n");
	}
	printf("\n");
	printf("Time %0.4f\n", t);
}
